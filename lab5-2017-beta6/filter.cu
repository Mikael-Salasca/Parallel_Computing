#include "hip/hip_runtime.h"
// Lab 5, image filters with CUDA.

// Compile with a command-line similar to Lab 4:
// nvcc filter.cu -c -arch=sm_30 -o filter.o
  // g++ filter.o milli.c readppm.c -lGL -lm -lcuda -lcudart -L/usr/local/cuda/lib -lglut -o filter
// or (multicore lab)
// nvcc filter.cu -c -arch=sm_20 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -L/usr/local/cuda/lib64 -lcudart -lglut -o filter

// 2017-11-27: Early pre-release, dubbed "beta".
// 2017-12-03: First official version! Brand new lab 5 based on the old lab 6.
// Better variable names, better prepared for some lab tasks. More changes may come
// but I call this version 1.0b2.
// 2017-12-04: Two fixes: Added command-lines (above), fixed a bug in computeImages
// that allocated too much memory. b3
// 2017-12-04: More fixes: Tightened up the kernel with edge clamping.
// Less code, nicer result (no borders). Cleaned up some messed up X and Y. b4

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#ifdef __APPLE__
  #include <GLUT/glut.h>
  #include <OpenGL/gl.h>
#else
  #include <GL/glut.h>
#endif
#include "readppm.h"
#include "milli.h"

// Use these for setting shared memory size.
#define maxKernelSizeX 10
#define maxKernelSizeY 10

#define TILE_W 8
#define TILE_H 8
#define KS_X 3
#define KS_Y 3
#define VERSION 5// 1 naive, 2 shared, 3 shared sep, 4 gaussian 5x5 sep, 5 median


__managed__
int kernelsize_pad = 0;
__managed__
int BLOCK_W = 0;
__managed__
int BLOCK_H = 0;

__global__ void filter(unsigned char * image, unsigned char * out,
    const unsigned int imagesizex,
        const unsigned int imagesizey,
        const int kernelsizex,
          const int kernelsizey) {
    // map from blockIdx to pixel position
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int dy, dx;
    unsigned int sumR, sumG, sumB;

    int divby = (2 * kernelsizex + 1) * (2 * kernelsizey + 1); // Works for box filters only!

    if (x < imagesizex && y < imagesizey) // If inside image
    {
        // Filter kernel (simple box filter)
        sumR = 0;
        sumG = 0;
        sumB = 0;
        for (dy = -kernelsizey; dy <= kernelsizey; dy++)
            for (dx = -kernelsizex; dx <= kernelsizex; dx++) {
                // Use max and min to avoid branching!
                int yy = min(max(y + dy, 0), imagesizey - 1);
                int xx = min(max(x + dx, 0), imagesizex - 1);

                sumR += image[((yy) * imagesizex + (xx)) * 3 + 0];
                sumG += image[((yy) * imagesizex + (xx)) * 3 + 1];
                sumB += image[((yy) * imagesizex + (xx)) * 3 + 2];
            }
        out[(y * imagesizex + x) * 3 + 0] = sumR / divby;
        out[(y * imagesizex + x) * 3 + 1] = sumG / divby;
        out[(y * imagesizex + x) * 3 + 2] = sumB / divby;
    }
}

// Filter optimized with shared memory
__global__ void filter_shared(unsigned char * image, unsigned char * out,
    const unsigned int imagesizex,
        const unsigned int imagesizey,
          const int kernelsizex,
            const int kernelsizey) {

    // Statically allocated shared memory
    __shared__ int s_i[(TILE_W + 2*maxKernelSizeX) * (TILE_H + 2*maxKernelSizeY) * 3];

    int x = blockIdx.x * TILE_W + threadIdx.x - kernelsizex;
    int y = blockIdx.y * TILE_H + threadIdx.y - kernelsizey;

    // clamp to edge of image
    x = min(max(x, 0), imagesizex-1);
    y = min(max(y,0), imagesizey-1);

    int index = x + y * imagesizex;
    int b_index = threadIdx.y * blockDim.y + threadIdx.x;

    // eah thread copies its pixel of the block to shared memory
    s_i[b_index * 3 + 0] = image[index * 3 + 0];
    s_i[b_index * 3 + 1] = image[index * 3 + 1];
    s_i[b_index * 3 + 2] = image[index * 3 + 2];

    __syncthreads();

    unsigned int sumR, sumG, sumB;

    int divby = (2 * kernelsizex + 1) * (2 * kernelsizey + 1); // Works for box filters only!

    int dy, dx;

    // Only threads inside the apron will write results
    if ((threadIdx.x >= kernelsizex)
          && (threadIdx.x < BLOCK_W - kernelsizex)
              && (threadIdx.y >= kernelsizey)
                && (threadIdx.y < BLOCK_H - kernelsizey))
    {
        // Filter kernel (simple box filter)
        sumR = 0;
        sumG = 0;
        sumB = 0;
        for (dy = -kernelsizey; dy <= kernelsizey; ++dy) {
          for (dx = -kernelsizex; dx <= kernelsizex; ++dx) {
                sumR += s_i[(b_index + dy*blockDim.x + dx) * 3 + 0];
                sumG += s_i[(b_index + dy*blockDim.x + dx) * 3 + 1];
                sumB += s_i[(b_index + dy*blockDim.x + dx) * 3 + 2];
            }
        }
        out[index * 3 + 0] = sumR / divby;
        out[index * 3 + 1] = sumG / divby;
        out[index * 3 + 2] = sumB / divby;
    } // end if
} // end filter shared

// Filter GAUSS 5x5
__global__ void filter_gauss_horizontal(unsigned char * image, unsigned char * out,
    const unsigned int imagesizex,
        const unsigned int imagesizey,
          const int kernelsizex,
            const int kernelsizey) {

    // Statically allocated shared memory
    __shared__ int s_i[(TILE_W + 2*maxKernelSizeX) * (TILE_H + 2*maxKernelSizeY) * 3];

    int x = blockIdx.x * TILE_W + threadIdx.x - kernelsizex;
    int y = blockIdx.y * TILE_H + threadIdx.y - kernelsizey;

    // clamp to edge of image
    x = min(max(x, 0), imagesizex-1);
    y = min(max(y,0), imagesizey-1);

    int index = x + y * imagesizex;
    int b_index = threadIdx.y * blockDim.y + threadIdx.x;

    // eah thread copies its pixel of the block to shared memory
    s_i[b_index * 3 + 0] = image[index * 3 + 0];
    s_i[b_index * 3 + 1] = image[index * 3 + 1];
    s_i[b_index * 3 + 2] = image[index * 3 + 2];

    __syncthreads();

    unsigned int sumR, sumG, sumB;

    int gauss_weight[5] = {1,4,6,4,1};
    int gauss_index;
    int divby = 16; // Works for 5x5 gauss

    int dy, dx;

    // Only threads inside the apron will write results
    if ((threadIdx.x >= kernelsizex)
          && (threadIdx.x < BLOCK_W - kernelsizex)
              && (threadIdx.y >= kernelsizey)
                && (threadIdx.y < BLOCK_H - kernelsizey))
    {
        // Filter kernel (simple box filter)
        sumR = 0;
        sumG = 0;
        sumB = 0;
        gauss_index = 0;

        for (dy = -kernelsizey; dy <= kernelsizey; ++dy) {
          for (dx = -kernelsizex; dx <= kernelsizex; ++dx) {
                sumR += s_i[(b_index + dy*blockDim.x + dx) * 3 + 0] * gauss_weight[gauss_index];
                sumG += s_i[(b_index + dy*blockDim.x + dx) * 3 + 1] * gauss_weight[gauss_index];
                sumB += s_i[(b_index + dy*blockDim.x + dx) * 3 + 2] * gauss_weight[gauss_index];
            }
            gauss_index++;
        }
        out[index * 3 + 0] = sumR / divby;
        out[index * 3 + 1] = sumG / divby;
        out[index * 3 + 2] = sumB / divby;
    } // end if

}

// Filter GAUSS 5x5
__global__ void filter_gauss_vertical(unsigned char * image, unsigned char * out,
    const unsigned int imagesizex,
        const unsigned int imagesizey,
          const int kernelsizex,
            const int kernelsizey) {

    // Statically allocated shared memory
    __shared__ int s_i[(TILE_W + 2*maxKernelSizeX) * (TILE_H + 2*maxKernelSizeY) * 3];

    int x = blockIdx.x * TILE_W + threadIdx.x - kernelsizex;
    int y = blockIdx.y * TILE_H + threadIdx.y - kernelsizey;

    // clamp to edge of image
    x = min(max(x, 0), imagesizex-1);
    y = min(max(y,0), imagesizey-1);

    int index = x + y * imagesizex;
    int b_index = threadIdx.y * blockDim.y + threadIdx.x;

    // eah thread copies its pixel of the block to shared memory
    s_i[b_index * 3 + 0] = image[index * 3 + 0];
    s_i[b_index * 3 + 1] = image[index * 3 + 1];
    s_i[b_index * 3 + 2] = image[index * 3 + 2];

    __syncthreads();

    unsigned int sumR, sumG, sumB;

    int gauss_weight[5] = {1,4,6,4,1};
    int gauss_index;
    int divby = 16; // Works for 5x5 gauss

    int dy, dx;

    // Only threads inside the apron will write results
    if ((threadIdx.x >= kernelsizex)
          && (threadIdx.x < BLOCK_W - kernelsizex)
              && (threadIdx.y >= kernelsizey)
                && (threadIdx.y < BLOCK_H - kernelsizey))
    {
        // Filter kernel (simple box filter)
        sumR = 0;
        sumG = 0;
        sumB = 0;
        gauss_index = 0;

          for (dx = -kernelsizex; dx <= kernelsizex; ++dx) {
                sumR += s_i[(b_index + dy*blockDim.x + dx) * 3 + 0] * gauss_weight[gauss_index];
                sumG += s_i[(b_index + dy*blockDim.x + dx) * 3 + 1] * gauss_weight[gauss_index];
                sumB += s_i[(b_index + dy*blockDim.x + dx) * 3 + 2] * gauss_weight[gauss_index];
                ++gauss_index;
            }
        out[index * 3 + 0] = sumR / divby;
        out[index * 3 + 1] = sumG / divby;
        out[index * 3 + 2] = sumB / divby;
    } // end if
}

__device__ void exchange(unsigned int *i, unsigned int *j) {
     int k;
     k = *i;
     *i = *j;
     *j = k;
}

__global__ void filter_median_horizontal(unsigned char * image, unsigned char * out,
    const unsigned int imagesizex,
        const unsigned int imagesizey,
          const int kernelsizex,
            const int kernelsizey) {

    // Statically allocated shared memory
    __shared__ int s_i[(TILE_W + 2*maxKernelSizeX) * (TILE_H + 2*maxKernelSizeY) * 3];

    int x = blockIdx.x * TILE_W + threadIdx.x - kernelsizex;
    int y = blockIdx.y * TILE_H + threadIdx.y - kernelsizey;

    // clamp to edge of image
    x = min(max(x, 0), imagesizex-1);
    y = min(max(y,0), imagesizey-1);

    int index = x + y * imagesizex;
    int b_index = threadIdx.y * blockDim.y + threadIdx.x;

    // eah thread copies its pixel of the block to shared memory
    s_i[b_index * 3 + 0] = image[index * 3 + 0];
    s_i[b_index * 3 + 1] = image[index * 3 + 1];
    s_i[b_index * 3 + 2] = image[index * 3 + 2];

    __syncthreads();

    unsigned int arr_R[maxKernelSizeX*2+1], arr_G[maxKernelSizeX*2+1], arr_B[maxKernelSizeX*2+1];
    unsigned int arr_index;

    int dy, dx;

    // Only threads inside the apron will write results
    if ((threadIdx.x >= kernelsizex)
          && (threadIdx.x < BLOCK_W - kernelsizex)
              && (threadIdx.y >= kernelsizey)
                && (threadIdx.y < BLOCK_H - kernelsizey))
    {
        arr_index = 0;
        for (dy = -kernelsizey; dy <= kernelsizey; ++dy) {
          for (dx = -kernelsizex; dx <= kernelsizex; ++dx) {
                arr_R[arr_index] = s_i[(b_index + dy*blockDim.x + dx) * 3 + 0];
                arr_G[arr_index] = s_i[(b_index + dy*blockDim.x + dx) * 3 + 1];
                arr_B[arr_index] = s_i[(b_index + dy*blockDim.x + dx) * 3 + 2];
            }
            arr_index++;
        }

        // sort arrays
        unsigned int i,j,k;

        //if (threadIdx.x == 0) {

          for (k=2;k<=arr_index;k=2*k) // Outer loop, double size for each step
          {
            for (j=k>>1;j>0;j=j>>1) // Inner loop, half size for each step
            {
              for (i=0;i<arr_index;i++) // Loop over ARR
              {
                int ixj=i^j; // Calculate indexing!
                if ((ixj)>i)
                {
                  if ((i&k)==0 && arr_R[i]>arr_R[ixj]) exchange(&arr_R[i],&arr_R[ixj]);
                  if ((i&k)!=0 && arr_R[i]<arr_R[ixj]) exchange(&arr_R[i],&arr_R[ixj]);

                  if ((i&k)==0 && arr_G[i]>arr_G[ixj]) exchange(&arr_G[i],&arr_G[ixj]);
                  if ((i&k)!=0 && arr_G[i]<arr_G[ixj]) exchange(&arr_G[i],&arr_G[ixj]);

                  if ((i&k)==0 && arr_B[i]>arr_B[ixj]) exchange(&arr_B[i],&arr_B[ixj]);
                  if ((i&k)!=0 && arr_B[i]<arr_B[ixj]) exchange(&arr_B[i],&arr_B[ixj]);
                }
              }
            }
          }
        //}

        //__syncthreads();

        out[index * 3 + 0] = arr_R[(arr_index-1)/2];
        out[index * 3 + 1] = arr_G[(arr_index-1)/2];
        out[index * 3 + 2] = arr_B[(arr_index-1)/2];
    } // end if
} // end filter median horizontal

// Filter optimized with shared memory
__global__ void filter_median_vertical(unsigned char * image, unsigned char * out,
    const unsigned int imagesizex,
        const unsigned int imagesizey,
          const int kernelsizex,
            const int kernelsizey) {

    // Statically allocated shared memory
    __shared__ int s_i[(TILE_W + 2*maxKernelSizeX) * (TILE_H + 2*maxKernelSizeY) * 3];

    int x = blockIdx.x * TILE_W + threadIdx.x - kernelsizex;
    int y = blockIdx.y * TILE_H + threadIdx.y - kernelsizey;

    // clamp to edge of image
    x = min(max(x, 0), imagesizex-1);
    y = min(max(y,0), imagesizey-1);

    int index = x + y * imagesizex;
    int b_index = threadIdx.y * blockDim.y + threadIdx.x;

    // eah thread copies its pixel of the block to shared memory
    s_i[b_index * 3 + 0] = image[index * 3 + 0];
    s_i[b_index * 3 + 1] = image[index * 3 + 1];
    s_i[b_index * 3 + 2] = image[index * 3 + 2];

    __syncthreads();

    unsigned int arr_R[maxKernelSizeX*2+1], arr_G[maxKernelSizeX*2+1], arr_B[maxKernelSizeX*2+1];
    unsigned int arr_index;

    int dy, dx;

    // Only threads inside the apron will write results
    if ((threadIdx.x >= kernelsizex)
          && (threadIdx.x < BLOCK_W - kernelsizex)
              && (threadIdx.y >= kernelsizey)
                && (threadIdx.y < BLOCK_H - kernelsizey))
    {
        arr_index = 0;
        for (dy = -kernelsizey; dy <= kernelsizey; ++dy) {
          for (dx = -kernelsizex; dx <= kernelsizex; ++dx) {
                arr_R[arr_index] = s_i[(b_index + dy*blockDim.x + dx) * 3 + 0];
                arr_G[arr_index] = s_i[(b_index + dy*blockDim.x + dx) * 3 + 1];
                arr_B[arr_index] = s_i[(b_index + dy*blockDim.x + dx) * 3 + 2];
                arr_index++;
            }
        }

        // sort arrays
        unsigned int i,j,k;

        //if (threadIdx.x == 0) {

          for (k=2;k<=arr_index;k=2*k) // Outer loop, double size for each step
          {
            for (j=k>>1;j>0;j=j>>1) // Inner loop, half size for each step
            {
              for (i=0;i<arr_index;i++) // Loop over ARR
              {
                int ixj=i^j; // Calculate indexing!
                if ((ixj)>i)
                {
                  if ((i&k)==0 && arr_R[i]>arr_R[ixj]) exchange(&arr_R[i],&arr_R[ixj]);
                  if ((i&k)!=0 && arr_R[i]<arr_R[ixj]) exchange(&arr_R[i],&arr_R[ixj]);

                  if ((i&k)==0 && arr_G[i]>arr_G[ixj]) exchange(&arr_G[i],&arr_G[ixj]);
                  if ((i&k)!=0 && arr_G[i]<arr_G[ixj]) exchange(&arr_G[i],&arr_G[ixj]);

                  if ((i&k)==0 && arr_B[i]>arr_B[ixj]) exchange(&arr_B[i],&arr_B[ixj]);
                  if ((i&k)!=0 && arr_B[i]<arr_B[ixj]) exchange(&arr_B[i],&arr_B[ixj]);
                }
              }
            }
          }
        //}

        //__syncthreads();

        out[index * 3 + 0] = arr_R[(arr_index-1)/2];
        out[index * 3 + 1] = arr_G[(arr_index-1)/2];
        out[index * 3 + 2] = arr_B[(arr_index-1)/2];
    } // end if
} // end filter median horizontal



// Global variables for image data
unsigned char * image, * pixels, * dev_bitmap, * dev_input;
unsigned int imagesizey, imagesizex; // Image size

////////////////////////////////////////////////////////////////////////////////
// main computation function
////////////////////////////////////////////////////////////////////////////////

void computeImages(int kernelsizex, int kernelsizey) {
    if (kernelsizex > maxKernelSizeX || kernelsizey > maxKernelSizeY) {
        printf("Kernel size out of bounds!\n");
        return;
    }

    pixels = (unsigned char * ) malloc(imagesizex * imagesizey * 3);
    hipMalloc((void ** ) & dev_input, imagesizex * imagesizey * 3);
    hipMemcpy(dev_input, image, imagesizey * imagesizex * 3, hipMemcpyHostToDevice);
    hipMalloc((void ** ) & dev_bitmap, imagesizex * imagesizey * 3);
    dim3 grid(imagesizex, imagesizey);
    hipEvent_t start;
    hipEvent_t end;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);

    filter << < grid, 1 >>> (dev_input, dev_bitmap, imagesizex, imagesizey,kernelsizex,kernelsizey); // Awful load balance

    hipDeviceSynchronize();
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);
    printf("Naive version - Elapsed time (ms): %f \n", time);    //	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    hipMemcpy(pixels, dev_bitmap, imagesizey * imagesizex * 3, hipMemcpyDeviceToHost);
    hipFree(dev_bitmap);
    hipFree(dev_input);
}

// ## compute image optimized with shared memory

inline unsigned int iDivUp( const unsigned int &a, const unsigned int &b ) { return ( a%b != 0 ) ? (a/b+1):(a/b); }

void computeImages_shared(int kernelsizex, int kernelsizey) {

  pixels = (unsigned char *) malloc(imagesizex*imagesizey*3);
	hipMalloc( (void**)&dev_input, imagesizex*imagesizey*3);
	hipMemcpy( dev_input, image, imagesizey*imagesizex*3, hipMemcpyHostToDevice );
	hipMalloc( (void**)&dev_bitmap, imagesizex*imagesizey*3);

   kernelsize_pad = max(kernelsizex,kernelsizey);
   BLOCK_W = TILE_W + 2*kernelsizex;
   BLOCK_H = TILE_H + 2*kernelsizey;


  const dim3 grid( iDivUp( imagesizex, TILE_W ), iDivUp( imagesizey, TILE_H ) );
  printf("gx=%d\n", grid.x);
  printf("gy=%d\n", grid.y);
  const dim3 threadBlock( BLOCK_W, BLOCK_W );
  printf("bx=%d\n", threadBlock.x);
  printf("by=%d\n", threadBlock.y);

  hipEvent_t start;
  hipEvent_t end;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);

	filter_shared<<<grid,threadBlock>>>(dev_input, dev_bitmap, imagesizey, imagesizex, kernelsizex, kernelsizey);

  hipDeviceSynchronize();
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&time, start, end);
  hipEventDestroy(start);
  hipEventDestroy(end);
  printf("Shared memory version - Elapsed time (ms): %f \n", time);

	hipDeviceSynchronize();
//	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
	hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
	hipFree( dev_bitmap );
	hipFree( dev_input );
} // end compute __shared__

void computeImages_shared_separable(int kernelsizex, int kernelsizey) {

  pixels = (unsigned char *) malloc(imagesizex*imagesizey*3);
	hipMalloc( (void**)&dev_input, imagesizex*imagesizey*3);
	hipMemcpy( dev_input, image, imagesizey*imagesizex*3, hipMemcpyHostToDevice );
	hipMalloc( (void**)&dev_bitmap, imagesizex*imagesizey*3);

   kernelsize_pad = max(kernelsizex,kernelsizey);
   BLOCK_W = TILE_W + 2*kernelsizex;
   BLOCK_H = TILE_H + 2*kernelsizey;


  const dim3 grid( iDivUp( imagesizex, TILE_W ), iDivUp( imagesizey, TILE_H ) );
  printf("gx=%d\n", grid.x);
  printf("gy=%d\n", grid.y);
  const dim3 threadBlock( BLOCK_W, BLOCK_W );
  printf("bx=%d\n", threadBlock.x);
  printf("by=%d\n", threadBlock.y);

  hipEvent_t start;
  hipEvent_t end;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);

	filter_shared<<<grid,threadBlock>>>(dev_input, dev_bitmap, imagesizey, imagesizex, kernelsizex, 1);
  hipDeviceSynchronize();
  filter_shared<<<grid,threadBlock>>>(dev_bitmap, dev_bitmap, imagesizey, imagesizex, 1, kernelsizey);

  hipDeviceSynchronize();
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&time, start, end);
  hipEventDestroy(start);
  hipEventDestroy(end);
  printf("Separable version - Elapsed time (ms): %f \n", time);

	hipDeviceSynchronize();
//	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
	hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
	hipFree( dev_bitmap );
	hipFree( dev_input );
}// end compute shared separable


void computeImages_gauss_separable(int kernelsizex, int kernelsizey) {

  pixels = (unsigned char *) malloc(imagesizex*imagesizey*3);
	hipMalloc( (void**)&dev_input, imagesizex*imagesizey*3);
	hipMemcpy( dev_input, image, imagesizey*imagesizex*3, hipMemcpyHostToDevice );
	hipMalloc( (void**)&dev_bitmap, imagesizex*imagesizey*3);

   kernelsize_pad = max(kernelsizex,kernelsizey);
   BLOCK_W = TILE_W + 2*2;
   BLOCK_H = TILE_H + 2*2;


  const dim3 grid( iDivUp( imagesizex, TILE_W ), iDivUp( imagesizey, TILE_H ) );
  printf("gx=%d\n", grid.x);
  printf("gy=%d\n", grid.y);
  const dim3 threadBlock( BLOCK_W, BLOCK_W );
  printf("bx=%d\n", threadBlock.x);
  printf("by=%d\n", threadBlock.y);

  hipEvent_t start;
  hipEvent_t end;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);

  filter_gauss_horizontal<<<grid,threadBlock>>>(dev_bitmap, dev_bitmap, imagesizey, imagesizex, 1, 2);
  hipDeviceSynchronize();
	filter_gauss_vertical<<<grid,threadBlock>>>(dev_input, dev_bitmap, imagesizey, imagesizex, 2, 1);

  hipDeviceSynchronize();
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&time, start, end);
  hipEventDestroy(start);
  hipEventDestroy(end);
  printf("Gaussian version - Elapsed time (ms): %f \n", time);

	hipDeviceSynchronize();
//	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
	hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
	hipFree( dev_bitmap );
	hipFree( dev_input );
}// end compute shared gaussian separable


void computeImages_median(int kernelsizex, int kernelsizey) {

  pixels = (unsigned char *) malloc(imagesizex*imagesizey*3);
	hipMalloc( (void**)&dev_input, imagesizex*imagesizey*3);
	hipMemcpy( dev_input, image, imagesizey*imagesizex*3, hipMemcpyHostToDevice );
	hipMalloc( (void**)&dev_bitmap, imagesizex*imagesizey*3);

   kernelsize_pad = max(kernelsizex,kernelsizey);
   BLOCK_W = TILE_W + 2*kernelsizex;
   BLOCK_H = TILE_H + 2*kernelsizey;


  const dim3 grid( iDivUp( imagesizex, TILE_W ), iDivUp( imagesizey, TILE_H ) );
  printf("gx=%d\n", grid.x);
  printf("gy=%d\n", grid.y);
  const dim3 threadBlock( BLOCK_W, BLOCK_W );
  printf("bx=%d\n", threadBlock.x);
  printf("by=%d\n", threadBlock.y);

  hipEvent_t start;
  hipEvent_t end;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);

  filter_median_horizontal<<<grid,threadBlock>>>(dev_bitmap, dev_bitmap, imagesizey, imagesizex, 1, kernelsizey);
  hipDeviceSynchronize();
  filter_median_vertical<<<grid,threadBlock>>>(dev_input, dev_bitmap, imagesizey, imagesizex, kernelsizex, 1);

  hipDeviceSynchronize();
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&time, start, end);
  hipEventDestroy(start);
  hipEventDestroy(end);
  printf("Median version - Elapsed time (ms): %f \n", time);

	hipDeviceSynchronize();
//	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
	hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
	hipFree( dev_bitmap );
	hipFree( dev_input );
} // end compute median


// Display images
void Draw() {
    // Dump the whole picture onto the screen.
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glClear(GL_COLOR_BUFFER_BIT);

    if (imagesizey >= imagesizex) { // Not wide - probably square. Original left, result right.
        glRasterPos2f(-1, -1);
        glDrawPixels(imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image);
        glRasterPos2i(0, -1);
        glDrawPixels(imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels);
    } else { // Wide image! Original on top, result below.
        glRasterPos2f(-1, -1);
        glDrawPixels(imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels);
        glRasterPos2i(-1, 0);
        glDrawPixels(imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image);
    }
    glFlush();
}

// Main program, inits
int main(int argc, char ** argv) {
    glutInit( & argc, argv);
    glutInitDisplayMode(GLUT_SINGLE | GLUT_RGBA);

    if (argc > 1)
        image = readppm(argv[1], (int * ) & imagesizex, (int * ) & imagesizey);
    else
        image = readppm((char * )
            "maskros512.ppm", (int * ) & imagesizex, (int * ) & imagesizey);

    if (imagesizey >= imagesizex)
        glutInitWindowSize(imagesizex * 2, imagesizey);
    else
        glutInitWindowSize(imagesizex, imagesizey * 2);
    glutCreateWindow("Lab 5");
    glutDisplayFunc(Draw);

    ResetMilli();

    printf("Kernel size X=%d\n",KS_X );
    printf("Kernel size Y=%d\n",KS_Y );
    printf("Kernel size =%d\n", ((KS_X*2+1) * (KS_Y*2+1)));

    int v = VERSION;
    switch (v) {
      case 1:
        computeImages(KS_X,KS_Y);
        break;
      case 2:
        computeImages_shared(KS_X,KS_Y);
        break;
      case 3:
        computeImages_shared_separable(KS_X,KS_Y);
        break;
      case 4:
        computeImages_gauss_separable(2,2);
        break;
      case 5:
        computeImages_median(KS_X,KS_Y);
        break;
      default:
        computeImages(KS_X,KS_Y);
        break;
    }


    // You can save the result to a file like this:
    //	writeppm("out.ppm", imagesizey, imagesizex, pixels);

    glutMainLoop();
    return 0;
}
