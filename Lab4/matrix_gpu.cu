#include "hip/hip_runtime.h"
// Matrix addition, CPU version
// nvcc matrix_gpu.cu milli.c -L /usr/local/cuda/lib -lcudart -o matrix

#include <stdio.h>
#include "milli.h"

__global__
void add_matrix_gpu(float *a, float *b, float *c, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int index = i + j*N;
	c[index] = a[index] + b[index];
}

void add_matrix_cpu(float *a, float *b, float *c, int N) {
	int index;

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			index = i + j*N;
			c[index] = a[index] + b[index];
		}
}

#define BLOCK_X_SIZE 16
#define BLOCK_Y_SIZE 16

#define N 8192


int main() {
	float *a = new float[N*N];
	float *b = new float[N*N];
	float *c = new float[N*N];

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}

		const int size = N*N*sizeof(float);

		//GPU
		float *a_gpu;
		float *b_gpu;
		float *c_gpu;
		hipEvent_t start_event, later_event;
		hipEventCreate(&start_event);
		hipEventCreate(&later_event);

		hipMalloc((void**)&a_gpu, size);
		hipMalloc((void**)&b_gpu, size);
		hipMalloc((void**)&c_gpu, size);

		hipMemcpy(a_gpu,(void*)a, size, hipMemcpyHostToDevice);
		hipMemcpy(b_gpu,(void*)b, size, hipMemcpyHostToDevice);

		dim3 threadsPerBlock(BLOCK_X_SIZE, BLOCK_Y_SIZE); // 16*16, 256 threads
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

		hipEventRecord(start_event, 0);

		add_matrix_gpu<<<numBlocks, threadsPerBlock>>>(a_gpu,b_gpu,c_gpu,N);

		hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

		hipDeviceSynchronize();
		hipMemcpy( c, c_gpu, size, hipMemcpyDeviceToHost);
		hipFree( a_gpu );
		hipFree( b_gpu );
		hipFree( c_gpu );

		hipEventRecord(later_event, 0);
	  hipEventSynchronize(later_event);
		float time = 0.0;
	  hipEventElapsedTime(&time, start_event, later_event);

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)	{
			// printf("%0.2f ", c[i+j*N]);
		}
		// printf("\n");
	}
	printf("N: %d \n", N);
	printf("Block size: %d \n", BLOCK_X_SIZE);
	printf("Nb blocks: %d \n", N / threadsPerBlock.x);
	printf("GPU - Elapsed time (ms): %f \n", time);

	// CPU
	int start = GetMicroseconds();
	add_matrix_cpu(a, b, c, N);
	float end = (GetMicroseconds() - (float)start ) / 1000;

	for (int i = 0; i < N; i++)	{
		for (int j = 0; j < N; j++)	{
			//printf("%0.2f ", c[i+j*N]);
		}
		//printf("\n");
	}

	printf("CPU - Elapsed Time (ms): %f\n ", end);

	delete[] a;
	delete[] b;
	delete[] c;

	printf("done\n");

	return EXIT_SUCCESS;

}
