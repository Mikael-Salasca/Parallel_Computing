// Matrix addition, CPU version
// nvcc matrix_gpu.cu -L /usr/local/cuda/lib -lcudart -o matrix_gpu


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void add_matrix(float *a, float *b, float *c, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int index = i + j*N;
	c[index] = a[index] + b[index];
}

#define BLOCK_SIZE 16
#define N 32


int main() {
	float a[N*N];
	float b[N*N];
	float c[N*N];
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}

		const int size = N*N*sizeof(float);

		float *a_gpu;
		float *b_gpu;
		float *c_gpu;
		hipEvent_t start_event, later_event;
		hipEventCreate(&start_event);
		hipEventCreate(&later_event);

		hipMalloc((void**)&a_gpu, size);
		hipMalloc((void**)&b_gpu, size);
		hipMalloc((void**)&c_gpu, size);

		hipMemcpy(a_gpu,(void*)a, size, hipMemcpyHostToDevice);
		hipMemcpy(b_gpu,(void*)b, size, hipMemcpyHostToDevice);
		dim3 dimGrid(N/BLOCK_SIZE, N/BLOCK_SIZE);
		dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
		hipEventRecord(start_event, 0);
		add_matrix<<<dimGrid, dimBlock>>>(a_gpu,b_gpu,c_gpu,N);

		hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

		hipDeviceSynchronize();
		hipMemcpy( c, c_gpu, size, hipMemcpyDeviceToHost);
		hipFree( a_gpu );
		hipFree( b_gpu );
		hipFree( c_gpu );

		hipEventRecord(later_event, 0);
	  hipEventSynchronize(later_event);
		float time = 0.0;
	  hipEventElapsedTime(&time, start_event, later_event);

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%0.2f ", c[i+j*N]);
		}
		printf("\n");
	}

	printf("done\n");

	printf("Elapsed time: %f \n", time/1000);

	return EXIT_SUCCESS;

}
